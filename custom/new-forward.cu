#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


#define TILE_WIDTH 16
#define MASK_WIDTH 7
#define MASK_RADIUS MASK_WIDTH / 2
#define SHARE_WIDTH TILE_WIDTH + MASK_RADIUS * 2


__constant__ float Mc[3136];

__global__ void conv_forward_kernel(float *y, const float *x, const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) Mc[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here

    const unsigned int W_grid  = ceil(W_out / (float) TILE_WIDTH);

    // int b = blockIdx.z * TILE_WIDTH + threadIdx.z;

    int ty = threadIdx.y;
    int tx = threadIdx.x;

    int b = blockIdx.x;
    int m = blockIdx.y;

    int h = (blockIdx.z / W_grid) * blockDim.y + ty;
    int w = (blockIdx.z % W_grid) * blockDim.x + tx;

    __shared__ float tile[4][SHARE_WIDTH][SHARE_WIDTH];

    if(w >= 0 && w < W && h >=0 && h < H){
        for (int c = 0 ; c < C; ++c)
            tile[c][ty][tx] = x4d(b, c, h, w);
    }
    else{
        for (int c = 0 ; c < C; ++c)
            tile[c][ty][tx] = 0.0f;
    }

    if(tx < K - 1){
         int temp_x =  w + TILE_WIDTH;

        if(temp_x >= 0 && temp_x < W && h >= 0 && h < H ){
            for (int c = 0 ; c < C; ++c)
                tile[c][ty][tx + TILE_WIDTH] = x4d(b, c, h, temp_x);
        }
        else{
            
            for (int c = 0 ; c < C; ++c)
                tile[c][ty][tx + TILE_WIDTH] = 0.0f;
        }
    }
    if(ty < K - 1){
         int temp_y =  h + TILE_WIDTH;

        if( temp_y >= 0 && temp_y < H && w >= 0 && w < W){
            for (int c = 0 ; c < C; ++c)
                tile[c][ty+ TILE_WIDTH][tx ] = x4d(b, c, temp_y, w);
        }
        else{
            
            for (int c = 0 ; c < C; ++c)
                tile[c][ty+ TILE_WIDTH][tx ] = 0.0f;
        }
    }

    if(tx < K - 1 && ty < K - 1){
        int temp_x =  w + TILE_WIDTH;
        int temp_y = h + TILE_WIDTH;

        if(temp_y >= 0 && temp_y < H && temp_x >= 0 && temp_x < W){
            for (int c = 0 ; c < C; ++c)
                tile[c][ty + TILE_WIDTH][tx + TILE_WIDTH] = x4d(b, c, temp_y, temp_x);
        }
        else{
            for (int c = 0 ; c < C; ++c)
                tile[c][ty + TILE_WIDTH][tx + TILE_WIDTH] = 0.0f;
        }
    }
     __syncthreads();

    
    if (w < W_out && h < H_out){
        float convolution = 0.0f;
        for (int c = 0; c < C; ++c)
        {
            for (int y_ = 0; y_ < K; ++y_)
            {
                for (int x_ = 0; x_ < K; ++x_)
                {
                    convolution +=  k4d(m, c, y_, x_) * tile[c][ty + y_][tx + x_];
                }
            }
        }

        y4d(b, m, h, w)  = convolution;
    }

#undef y4d
#undef x4d
#undef k4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_y, const float *host_x, const float *host_k, float **device_y_ptr, float **device_x_ptr, float **device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    const unsigned int sizeX = B * C * H * W * sizeof(float);
    const unsigned int sizeY = B * M * H_out * W_out * sizeof(float);
    const unsigned int sizeK = M * C * K * K * sizeof(float);

    hipMalloc((void **)device_x_ptr, sizeX);
    hipMalloc((void **)device_y_ptr, sizeY);

    hipMemcpy(*device_x_ptr, host_x, sizeX, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(Mc), host_k, sizeK, 0 , hipMemcpyHostToDevice);

    std::cout << sizeK << " " << C << " " << K << std::endl;
}


__host__ void GPUInterface::conv_forward_gpu(float *device_y, const float *device_x, const float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Set the kernel dimensions and call the kernel
    const unsigned int H_out = H - K + 1;
    const unsigned int W_out = W - K + 1;

    const unsigned int W_grid  = ceil(W_out / (float) TILE_WIDTH);
    const unsigned int H_grid = ceil(H_out / (float) TILE_WIDTH);

    const unsigned Y = H_grid * W_grid;

    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 gridDim( B, M, Y);

    conv_forward_kernel<<<gridDim,  blockDim >>>(device_y, device_x,  B, M , C, H, W, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_y, float *device_y, float *device_x, float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    // Copy the output back to host
    const unsigned int sizeY = B * M * H_out * W_out * sizeof(float);
    hipMemcpy(host_y, device_y, sizeY, hipMemcpyDeviceToHost);

    // Free device memory

    hipFree(device_x);
    hipFree(device_y);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
