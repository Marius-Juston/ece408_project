#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


#define TILE_WIDTH 16

__constant__ float Mc[3136];

__global__ void conv_forward_kernel(float *y, const float *x, const int B, const int M, const int C, const int H, const int W, const int K)
__global__ void conv_forward_kernel_2(float *y, const float *x, const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) Mc[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here

    const unsigned int ty = threadIdx.y;
    const unsigned int tx = threadIdx.x;

    const unsigned int b = blockIdx.z;

    // const unsigned int row = blockIdx.y * TILE_WIDTH_2 + ty;
    const unsigned int col = blockIdx.x * TILE_WIDTH_2 + tx;

    // __shared__ float rowShared[TILE_WIDTH_2][TILE_WIDTH_2];
    __shared__ float colShared[TILE_WIDTH_2][TILE_WIDTH_2];

    const unsigned int numBlocks = ceil(C * K * K / (float) TILE_WIDTH_2);

    const unsigned int W_BASE = C * K * K;

    float sum = 0.0;

    const bool compute = row < M && col < W_out * H_out; 

    const unsigned int X_h = col / W_out;
    const unsigned int X_w = col % W_out;
    
    for (int i = 0; i < numBlocks; ++i){
        const unsigned int tileCol = i * TILE_WIDTH_2 + tx; // For the kernel
        const unsigned int tileRow = i * TILE_WIDTH_2 + ty; // for the input

        // input matrix shared memeory

        if(tileRow < W_BASE && col < H_out * W_out){
            const unsigned int temp = tileRow % (K * K) ;
            const unsigned int X_p =  temp / K;
            const unsigned int X_q = temp % K;

            colShared[ty][tx] = x4d(b, tileRow / (K * K), X_h + X_p , X_w + X_q);
        }else{
            colShared[ty][tx] = 0.0f;            
        }

        // if(tileCol < W_BASE && row < M){
        //     const unsigned int K_c = tileCol / (K * K);

        //     const unsigned int temp = (tileCol % (K * K)); 
        //     const unsigned int K_h =  temp / K;
        //     const unsigned int K_w = temp % K;

        //     rowShared[ty][tx] = k4d(row , K_c, K_h, K_w);
        // }else{
        //     rowShared[ty][tx] = 0.0f;
        // }

        __syncthreads();

        if(compute){


            int tileKernel;
            int K_c;
            int temp;
            int K_h;
            int K_w;
            for(int k = 0; k < TILE_WIDTH_2; ++k){
                tileKernel = i * TILE_WIDTH_2 + k;
                K_c = tileKernel / (K * K);
                temp = (tileKernel % (K * K)); 
                K_h =  temp / K;
                K_w = temp % K;

                sum += colShared[k][tx] * k4d(row , K_c, K_h, K_w);            
                // sum += colShared[k][tx] * rowShared[ty][k];  
            }
        }
        __syncthreads();
    }

    if(compute){        
        y4d(b, row, X_h, X_w) = sum;
    }


#undef y4d
#undef x4d
#undef k4d
}
	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_y, const float *host_x, const float *host_k, float **device_y_ptr, float **device_x_ptr, float **device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    const unsigned int sizeX = B * C * H * W * sizeof(float);
    const unsigned int sizeY = B * M * H_out * W_out * sizeof(float);
    const unsigned int sizeK = M * C * K * K * sizeof(float);

    hipMalloc((void **)device_x_ptr, sizeX);
    hipMalloc((void **)device_y_ptr, sizeY);

    hipMemcpy(*device_x_ptr, host_x, sizeX, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(Mc), host_k, sizeK, 0 , hipMemcpyHostToDevice);

    std::cout << sizeK << " " << C << " " << K << std::endl;
}


__host__ void GPUInterface::conv_forward_gpu(float *device_y, const float *device_x, const float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Set the kernel dimensions and call the kernel
    const unsigned int H_out = H - K + 1;
    const unsigned int W_out = W - K + 1;

    const unsigned int M_ = ceil(M / (float) TILE_WIDTH);

    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 gridDim( ceil((W_out * H_out) / (float) ( TILE_WIDTH)), M_, B);

    conv_forward_kernel<<<gridDim,  blockDim >>>(device_y, device_x,  B, M , C, H, W, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_y, float *device_y, float *device_x, float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    // Copy the output back to host
    const unsigned int sizeY = B * M * H_out * W_out * sizeof(float);
    hipMemcpy(host_y, device_y, sizeY, hipMemcpyDeviceToHost);

    // Free device memory

    hipFree(device_x);
    hipFree(device_y);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
